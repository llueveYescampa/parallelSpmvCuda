#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "dataDef.h"
#include "real.h"

__global__ 
void spmv0(real *__restrict__ y, 
           real *__restrict__ x, 
           real *__restrict__ val, 
           int  *__restrict__ row_ptr, 
           int  *__restrict__ col_idx, 
           const int nRows
           )
{    
    const unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (row < nRows)  {
        real dot = (real) 0;
        for (int col = row_ptr[row]; col < row_ptr[row+1]; ++col ) {
            dot += (val[col] * x[col_idx[col]]);
        } // end for //
        y[row] = dot;
    } // end if //
} // end of spmv0() //

__global__ 
void spmv1(real *__restrict__ y, 
           real *__restrict__ x, 
           real *__restrict__ val, 
           int *__restrict__  row_ptr, 
           int *__restrict__  col_idx, 
           int nRows)
{    
    extern __shared__ real temp[];
    temp[threadIdx.x] = (real) 0;

    const unsigned int row = blockIdx.x;
    
    for (unsigned int col=row_ptr[row]+threadIdx.x; col<row_ptr[row+1]; col+=blockDim.x) {
        temp[threadIdx.x] += (val[col] * x[col_idx[col]]);
    } // end for //
    __syncthreads();
    
    // local reduction per block
    for (unsigned int next = blockDim.x/2; next > 0; next >>= 1 ) {
        if (threadIdx.x < next) {
            temp[threadIdx.x]+=temp[threadIdx.x+next];
        } // end if // 
        __syncthreads();
    } // end for //

    if (threadIdx.x == 0) {
        y[blockIdx.x] = temp[0];
    } // end if //   
} // end of spmv1() //
