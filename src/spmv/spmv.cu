#include "hip/hip_runtime.h"
#include <stdio.h>
#include "real.h"

#ifdef DOUBLE
    extern texture<int2> xTex;
    extern texture<int2> valTex;
#else
    extern texture<float> xTex;
    extern texture<float> valTex;
#endif

#ifdef DOUBLE
    static __inline__ __device__ 
    double fetch_real(texture<int2> t, int i)
    {
	    int2 v = tex1Dfetch(t,i);
	    return __hiloint2double(v.y, v.x);
    } // end of fetch_real() //
#else
    static __inline__ __device__ 
    float fetch_real(texture<float> t, int i)
    {
	    return tex1Dfetch(t,i);
    } // end of fetch_double() //
#endif

__global__ 
void spmv(real *__restrict__ y, 
           //real *__restrict__ x, 
           //real *__restrict__ val, 
           int  *__restrict__ row_ptr, 
           int  *__restrict__ col_idx, 
           const int nRows
          )
{   
    if (blockDim.y==1) { 
        const int row = blockIdx.x*blockDim.x + threadIdx.x;
        if (row < nRows)  {
            real dot = (real) 0.0;
            for ( int col = row_ptr[row]; col < row_ptr[row+1]; ++col ) {
                //dot += (val[col] * x[col_idx[col]]);
                dot += (fetch_real(valTex,col) * fetch_real( xTex, col_idx[col])); 
            } // end for //
            y[row] = dot;
        } // end if //
    } else {    
        extern __shared__ real temp[];
        const unsigned int row = blockIdx.x*blockDim.y + threadIdx.y;
        const unsigned int sharedMemIndx = blockDim.x*threadIdx.y + threadIdx.x;
        temp[sharedMemIndx] = (real) 0.0;

        if (row < nRows) {
            for (unsigned int col=row_ptr[row]+threadIdx.x; col < row_ptr[row+1]; col+=blockDim.x) {
                //temp[threadIdx.x] += (val[col] * x[col_idx[col]]);
                temp[ sharedMemIndx] += (fetch_real(valTex,col) * fetch_real( xTex, col_idx[col]));
            } // end for //
            __syncthreads();

            if (blockDim.x == 64) {
                if (threadIdx.x<32) temp[sharedMemIndx] += temp[sharedMemIndx + 32];
                __syncthreads();
            } // end if //
          
            // unrolling warp 
            if (threadIdx.x < 16) {
                volatile real *temp1 = temp;
                temp1[sharedMemIndx] += temp1[sharedMemIndx + 16];
                temp1[sharedMemIndx] += temp1[sharedMemIndx + 8];
                temp1[sharedMemIndx] += temp1[sharedMemIndx + 4];
                temp1[sharedMemIndx] += temp1[sharedMemIndx + 2];
                temp1[sharedMemIndx] += temp1[sharedMemIndx + 1];
            } // end if //

            if ((sharedMemIndx % blockDim.x)  == 0) {
                y[row] = temp[sharedMemIndx];
            } // end if //   
        } // end if
    } // end if //
    
} // end of spmv() //
