#include "hip/hip_runtime.h"
#include <stdio.h>
#include "real.h"


#ifdef DOUBLE
    extern texture<int2> xTex;
    extern texture<int2> valTex;
#else
    extern texture<float> xTex;
    extern texture<float> valTex;
#endif

#ifdef DOUBLE
    static __inline__ __device__ 
    double fetch_real(texture<int2> t, int i)
    {
	    int2 v = tex1Dfetch(t,i);
	    return __hiloint2double(v.y, v.x);
    } // end of fetch_real() //
#else
    static __inline__ __device__ 
    float fetch_real(texture<float> t, int i)
    {
	    return tex1Dfetch(t,i);
    } // end of fetch_double() //
#endif

__global__ 
void spmv0(real *__restrict__ y, 
           //real *__restrict__ x, 
           //real *__restrict__ val, 
           int  *__restrict__ row_ptr, 
           int  *__restrict__ col_idx, 
           const int nRows
          )
{    
    const unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (row < nRows)  {
        real dot = (real) 0;
        for (int col = row_ptr[row]; col < row_ptr[row+1]; ++col ) {
            //dot += (val[col] * x[col_idx[col]]);
            dot += (fetch_real(valTex,col) * fetch_real( xTex, col_idx[col])); 
        } // end for //
        y[row] = dot;
    } // end if //
} // end of spmv0() //

__global__ 
void spmv1(real *__restrict__ y, 
           //real *__restrict__ x, 
           //real *__restrict__ val, 
           int *__restrict__  row_ptr, 
           int *__restrict__  col_idx, 
           int nRows)
{    
    extern __shared__ real temp[];
    temp[threadIdx.x] = (real) 0;

    const int row = blockIdx.x;
    
    for (int col=row_ptr[row]+threadIdx.x; col < row_ptr[row+1]; col+=blockDim.x) {
        //temp[threadIdx.x] += (val[col] * x[col_idx[col]]);
        temp[threadIdx.x] += (fetch_real(valTex,col) * fetch_real( xTex, col_idx[col]));
    } // end for //
    __syncthreads();
    
    // local reduction per block
    for (int next = blockDim.x/2; next > 0; next >>= 1 ) {
        if (threadIdx.x < next) {
            temp[threadIdx.x] += temp[threadIdx.x+next];
        } // end if // 
        __syncthreads();
    } // end for //

    if (threadIdx.x == 0) {
        y[blockIdx.x] = temp[0];
    } // end if //   
} // end of spmv1() //
